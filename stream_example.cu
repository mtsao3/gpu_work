/*https://cdac.in/index.aspx?id=ev_hpc_gpu-comp-nvidia-cuda-streams#hetr-cuda-prog-cuda-streams*/

#include <stdio.h> 
#include <time.h> 
#include <hip/hip_runtime.h> 

#define BLOCKSIZE 256
#define SIZEOFARRAY 1048576*4
#define KENERL_LOOP 400
 
// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

__global__ void arrayAddition(int *device_a, int *device_b, int *device_result, const int offset)
{

	int threadId = threadIdx.x + blockIdx.x * blockDim.x ;
	int index = threadId + offset;

	if (threadId < SIZEOFARRAY)
			for (int i =0; i < KENERL_LOOP; i++)
        device_result[index]= device_a[index]+device_b[index]; 
} 

__host__ void generate_rand_data(unsigned int * host_data_ptr)
{
        for(unsigned int i=0; i < SIZEOFARRAY; i++)
        {
                host_data_ptr[i] = (unsigned int) rand();
        }
}

__host__ void start_measure(hipEvent_t * start, hipEvent_t *stop){
        hipEventCreateWithFlags(start,0);
        hipEventCreateWithFlags(stop,0);
        hipEventRecord(*start, 0);
}

__host__ void stop_measure(hipEvent_t* start, hipEvent_t * stop, float &time) {
        hipEventRecord(*stop, 0);
        hipEventSynchronize(*stop);
        hipEventElapsedTime(&time, *start, *stop);
}

/* Check for safe return of all calls to the device */ 
int main ( int argc, char **argv ) 
{ 

	// Get cuda properties
  hipDeviceProp_t prop; 
  hipSetDevice(0);
  hipGetDeviceProperties( &prop, 0); 
  printf("maxThreadsPerBlock is %d \n", prop.maxThreadsPerBlock);

  // Allocate device and host memory
  const int num_streams = 4;
  const int stream_size = SIZEOFARRAY / num_streams;
  const int stream_bytes = stream_size * sizeof(int);
  const int num_bytes = SIZEOFARRAY * sizeof(int);
  int *host_a, *host_b, *host_result; 
  int *device_a, *device_b, *device_result; 

  checkCuda(hipMalloc( ( void**)& device_a, num_bytes)); 
  checkCuda(hipMalloc( ( void**)& device_b, num_bytes )); 
  checkCuda(hipMalloc( ( void**)& device_result, num_bytes)); 

  checkCuda(hipHostAlloc((void **)&host_a, num_bytes, hipHostMallocDefault));
  checkCuda(hipHostAlloc((void **)&host_b, num_bytes, hipHostMallocDefault));
  checkCuda(hipHostAlloc((void **)&host_result, num_bytes, hipHostMallocDefault));
  
  // Instantiate cuda events and streams
  hipEvent_t start, stop, start2, stop2;
  float elapsedTime, elapsedTime2; 

  // Create Streams
  hipStream_t orig;
  hipStream_t stream[num_streams];
  checkCuda( hipStreamCreate(&orig));
  for (int i = 0; i < num_streams; ++i)
    checkCuda( hipStreamCreate(&stream[i]) );

  // Instantiate host values
  for(int index = 0; index < SIZEOFARRAY; index++) 
  { 
  	host_a[index] = index;
  	host_b[index] = SIZEOFARRAY - index;
  } 

  // Run sequential version
  start_measure(&start, &stop);
  checkCuda(hipMemcpyAsync(device_a, host_a, num_bytes, hipMemcpyHostToDevice, orig)); 
	checkCuda(hipMemcpyAsync(device_b, host_b, num_bytes, hipMemcpyHostToDevice, orig)); 
  arrayAddition<<<SIZEOFARRAY/BLOCKSIZE, BLOCKSIZE>>>(device_a, device_b, device_result, 0);
  checkCuda(hipMemcpyAsync(host_result, device_result, num_bytes, hipMemcpyDeviceToHost, orig)); 
  stop_measure(&start, &stop, elapsedTime);

  // Run overlapped stream processing
  //		each stream processes portions of the data
  start_measure(&start2, &stop2);

  for (int i = 0; i < num_streams; ++i) {
	  int offset = i * stream_size;
  	checkCuda(hipMemcpyAsync(&device_a[offset], &host_a[offset], stream_bytes, hipMemcpyHostToDevice, stream[i])); 
  	checkCuda(hipMemcpyAsync(&device_b[offset], &host_b[offset], stream_bytes, hipMemcpyHostToDevice, stream[i]));
	}

	for (int i = 0; i < num_streams; ++i) {
	  int offset = i * stream_size;
	  arrayAddition<<<stream_size/BLOCKSIZE, BLOCKSIZE, 0, stream[i]>>>(device_a, device_b, device_result, offset);
	}

	for (int i = 0; i < num_streams; ++i) {
	  int offset = i * stream_size;
	  checkCuda(hipMemcpyAsync(&host_result[offset], &device_result[offset], stream_bytes, hipMemcpyDeviceToHost, stream[i])); 
	}
  stop_measure(&start2, &stop2, elapsedTime2);

  printf("\n Block size: %d \n", BLOCKSIZE);
  printf("\n Kernal loop size: %d \n", KENERL_LOOP); 
  printf("\n Size of array : %d \n", SIZEOFARRAY); 
  printf("\n Sequential Time taken: %3.1f ms \n", elapsedTime);
  printf("\n Streams Time taken: %3.1f ms \n", elapsedTime2);

  hipHostFree(host_a); 
  hipHostFree(host_b); 
  hipHostFree(host_result); 
  hipFree(device_a); 
  hipFree(device_b); 
  hipFree(device_result); 

  return 0; 
}
