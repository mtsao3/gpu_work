
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>


#define NUM_ELEMENTS 8192

// Non interleaved structure definition
typedef unsigned int ARRAY_MEMBER_T[NUM_ELEMENTS];
typedef struct {
	ARRAY_MEMBER_T a;
	ARRAY_MEMBER_T b;
	ARRAY_MEMBER_T c;
	ARRAY_MEMBER_T d;
} NON_INTERLEAVED_T;


// Multiply kernel
__global__ void multiply_kernel(
		NON_INTERLEAVED_T * const dest_ptr,
		NON_INTERLEAVED_T * const src_ptr,
		const unsigned int num_elements) {
	const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
	if(tid < num_elements)
	{
			dest_ptr->a[tid] *= src_ptr->a[tid];
			dest_ptr->b[tid] *= src_ptr->b[tid];
			dest_ptr->c[tid] *= src_ptr->c[tid];
			dest_ptr->d[tid] *= src_ptr->d[tid];
	}
}

int main(void)
{
  // Define structs
  int bytes = sizeof(NON_INTERLEAVED_T);
  NON_INTERLEAVED_T *x, *y, *x_pin, *y_pin, *d_x, *d_y;
  const unsigned int num_threads = 256;
	const unsigned int num_blocks = (NUM_ELEMENTS + (num_threads-1)) / num_threads;

    // Define measurement 
  hipEvent_t kernel_start, kernel_stop;
  hipEvent_t kernel_start1, kernel_stop1;
  hipEventCreateWithFlags(&kernel_start,0);
	hipEventCreateWithFlags(&kernel_stop,0);
  hipEventCreateWithFlags(&kernel_start1,0);
	hipEventCreateWithFlags(&kernel_stop1,0);

  // Allocate pageable memory
  x = (NON_INTERLEAVED_T*)malloc(bytes);
  y = (NON_INTERLEAVED_T*)malloc(bytes);
  
  // Allocate pinned memory
  hipHostMalloc((void**)&x_pin, bytes, hipHostMallocDefault);
  hipHostMalloc((void**)&y_pin, bytes, hipHostMallocDefault);
  
  // Allocate device memory
  hipMalloc(&d_x, bytes); 
  hipMalloc(&d_y, bytes);
  
  // Fill data
  float x_val = 3.0f;
  float y_val = 2.0f;
  for (int i = 0; i < NUM_ELEMENTS; i++) {
    x->a[i] = x_val;
    x->b[i] = x_val;
    x->c[i] = x_val;
    x->d[i] = x_val;
    y->a[i] = y_val;
    y->b[i] = y_val;
    y->c[i] = y_val;
    y->d[i] = y_val;
    x_pin->a[i] = x_val;
    x_pin->b[i] = x_val;
    x_pin->c[i] = x_val;
    x_pin->d[i] = x_val;
    y_pin->a[i] = y_val;
    y_pin->b[i] = y_val;
    y_pin->c[i] = y_val;
    y_pin->d[i] = y_val;
  }
  
  hipEventRecord(kernel_start, 0);

  // Copy pageable memory
  hipMemcpy(d_x, x, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, bytes, hipMemcpyHostToDevice);
  
  // Apply Kernel
  multiply_kernel<<<num_blocks, num_threads>>>(d_y, d_x, NUM_ELEMENTS);

  // Copy data back
  hipMemcpy(y, d_y, bytes, hipMemcpyDeviceToHost);
  
  // Display metrics
  hipEventRecord(kernel_stop, 0);
  hipEventSynchronize(kernel_stop);
	float delta = 0.0F;
	hipEventElapsedTime(&delta, kernel_start, kernel_stop);
  std::cout << "Pageable multiply took " << delta << std::endl;

  hipEventRecord(kernel_start1, 0);

  // Copy pinned memory
  hipMemcpy(d_x, x_pin, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y_pin, bytes, hipMemcpyHostToDevice);
  
  // Apply kernel
  multiply_kernel<<<num_blocks, num_threads>>>(d_y, d_x, NUM_ELEMENTS);
  
  // Copy memory back
  hipMemcpy(y_pin, d_y, bytes, hipMemcpyDeviceToHost);
  
  // Display metrics
  hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
  float delta1 = 0.0F;
	hipEventElapsedTime(&delta1, kernel_start1, kernel_stop1);
  std::cout << "Pinned multiply took " << delta1 << std::endl;
  
  // // Print some values for validation
  // for(int i = NUM_ELEMENTS-3; i < NUM_ELEMENTS; i++) {
  //   std::cout << "y_pin.a[" << i << "] = " << y_pin->a[i] << std::endl;
  //   std::cout << "y.a[" << i << "] = " << y->a[i] << std::endl;
  //   std::cout << "y_pin.b[" << i << "] = " << y_pin->b[i] << std::endl;
  //   std::cout << "y.b[" << i << "] = " << y->b[i] << std::endl;
  //   std::cout << "y_pin.c[" << i << "] = " << y_pin->c[i] << std::endl;
  //   std::cout << "y.c[" << i << "] = " << y->c[i] << std::endl;
  //   std::cout << "y_pin.d[" << i << "] = " << y_pin->d[i] << std::endl;
  //   std::cout << "y.d[" << i << "] = " << y->d[i] << std::endl;
  // }

  // House keeping
  hipEventDestroy(kernel_start);
	hipEventDestroy(kernel_stop);
  hipEventDestroy(kernel_start1);
	hipEventDestroy(kernel_stop1);
  hipHostFree(x_pin);
  hipHostFree(x_pin);
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
}

